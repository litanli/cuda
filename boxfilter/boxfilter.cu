#include "hip/hip_runtime.h"
#include <cmath>
#include <cassert>
#include <cstdint>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

// For png load and save
#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image.h"
#include "stb_image_write.h"

using namespace std;


// Apply a Box filter to blur an image.
// Note for FILTER_SIZE > 1, each thread should collaborate to load shared
// pixel values from shared memory. This kernel does not do so - making it a
// naive implementation that serves as a performance baseline.
__global__
void box_filter(uint8_t* in, uint8_t* out, int h, int w, int channels, 
                int filter_size) {
    
    int out_row = blockIdx.y * blockDim.y + threadIdx.y;
    int out_col = blockIdx.x * blockDim.x + threadIdx.x;

    if (out_row < h && out_col < w) {

        int pixel_val = 0;
        int pixel_count = 0;

    }
}


// Stub
void box_filter(int filter_size) {
    if (filter_size % 2 == 0) {
        cerr << "Filter size must be odd." << endl;
        exit(1);
    }
}